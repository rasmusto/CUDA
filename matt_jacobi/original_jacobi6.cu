
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

 __global__ void
jacobikernel( float* a, float* newa, float* lchange, int n, int m, float w0, float w1, float w2 )
{
    int ti = threadIdx.x;
    int tj = threadIdx.y;
    int i = blockIdx.x * blockDim.x + ti + 1;
    int j = blockIdx.y * blockDim.y + tj + 1;
    __shared__ float mychange[18*18];
    float mnewa, molda;


    mychange[tj*18+ti] = a[(j-1)*m+i-1];
    if( ti < 2 ) mychange[tj*18+ti+16] = a[(j-1)*m+i+15];
    if( tj < 2 ) mychange[(tj+16)*18+ti] = a[(j+15)*m+i-1];
    if( tj < 2 && ti < 2 ) mychange[(tj+16)*18+ti+16] = a[(j+15)*m+i+15];

    __syncthreads();

    molda = mychange[(tj+1)*18+(ti+1)];
    mnewa = w0*molda +
	    w1 * (mychange[(tj+1)*18+(ti  )] + mychange[(tj  )*18+(ti+1)] +
		  mychange[(tj+1)*18+(ti+2)] + mychange[(tj+2)*18+(ti+1)]) +
	    w2 * (mychange[(tj  )*18+(ti  )] + mychange[(tj+2)*18+(ti  )] +
		  mychange[(tj  )*18+(ti+2)] + mychange[(tj+2)*18+(ti+2)]);
    newa[j*m+i] = mnewa;
    __syncthreads();

    int ii = ti+blockDim.x*tj;
    mychange[ii] = fabsf( mnewa - molda );
    __syncthreads();
    int nn = blockDim.x * blockDim.y;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf( mychange[ii], mychange[ii+nn] );
	__syncthreads();
    }
    if( ii == 0 )
	lchange[blockIdx.x + gridDim.x*blockIdx.y] = mychange[0];
}

 __global__ void
reductionkernel( float* lchange, int n )
{
    __shared__ float mychange[256];
    float mych = 0.0f;
    int ii = threadIdx.x, m;
    if( ii < n ) mych = lchange[ii];
    m = blockDim.x;
    while( m <= n ){
	mych = fmaxf( mych, lchange[ii+m] );
	m += blockDim.x;
    }
    mychange[ii] = mych;
    __syncthreads();
    int nn = blockDim.x;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf(mychange[ii],mychange[ii+nn]);
	__syncthreads();
    }
    if( ii == 0 )
	lchange[0] = mychange[0];
}

static float sumtime;


void JacobiGPU( float* a, int n, int m, float w0, float w1, float w2, float tol )
{
    float change;
    int iters;
    size_t memsize;
    int bx, by, gx, gy;
    float *da, *dnewa, *lchange;
    hipEvent_t e1, e2;

    float changeCheck = 0, oldchange = 0;

    bx = 16;
    by = 16;
    gx = (n-2)/bx + ((n-2)%bx == 0?0:1);
    gy = (m-2)/by + ((m-2)%by == 0?0:1);

    sumtime = 0.0f;
    memsize = sizeof(float) * n * m;
    hipMalloc( &da, memsize );
    hipMalloc( &dnewa, memsize );
    hipMalloc( &lchange, gx * gy * sizeof(float) );
    hipEventCreate( &e1 );
    hipEventCreate( &e2 );

    dim3 block( bx, by );
    dim3 grid( gx, gy );

    iters = 0;
    hipMemcpy( da, a, memsize, hipMemcpyHostToDevice );
    hipMemcpy( dnewa, a, memsize, hipMemcpyHostToDevice );
    do{
	float msec;
	++iters;

	hipEventRecord( e1 );
	jacobikernel<<< grid, block >>>( da, dnewa, lchange, n, m, w0, w1, w2 );
	reductionkernel<<< 1, bx*by >>>( lchange, gx*gy );
	hipEventRecord( e2 );

	hipMemcpy( &change, lchange, sizeof(float), hipMemcpyDeviceToHost );
	hipEventElapsedTime( &msec, e1, e2 );
	sumtime += msec;
	float *ta;
	ta = da;
	da = dnewa;
	dnewa = ta;  
	//printf("iters = %d, change = %f\n", iters, change);
	if(change == oldchange)
	{
		changeCheck++;
	}
	oldchange = change;
	if(changeCheck > sqrt(m))
	{
		change = (tol - .01);
	}
	printf("iters = %d, change = %f, changeCheck = %f, oldchange = %f\n", iters, change, changeCheck, oldchange);

   }while( change > tol );
    printf( "JacobiGPU  converged in %d iterations to residual %f\n", iters, change );
    printf( "JacobiGPU  used %f seconds total\n", sumtime/1000.0f );
    hipMemcpy( a, dnewa, memsize, hipMemcpyDeviceToHost );
    hipFree( da );
    hipFree( dnewa );
    hipFree( lchange );
    hipEventDestroy( e1 );
    hipEventDestroy( e2 );
}

static void init( float* a, int n, int m )
{
    int i, j;
    memset( a, 0, sizeof(float) * n * m );
    /* boundary conditions */
    for( j = 0; j < n; ++j ){
	a[j*m+n-1] = j;
    }
    for( i = 0; i < m; ++i ){
	a[(n-1)*m+i] = i;
    }
    a[(n-1)*m+m-1] = m+n;
}

int main( int argc, char* argv[] )
{
    int n, m;
    float *a;
    struct timeval tt1, tt2;
    int ms;
    float fms;

    if( argc <= 1 ){
	fprintf( stderr, "%s sizen [sizem]\n", argv[0] );
	return 1;
    }

    n = atoi( argv[1] );
    if( n <= 0 ) n = 100;
    m = n;
    if( argc > 2 ){
	m = atoi( argv[2] );
	if( m <= 0 ) m = 100;
    }

    printf( "Jacobi %d x %d\n", n, m );

    a = (float*)malloc( sizeof(float) * n * m );
    
    init( a, n, m );
 
    gettimeofday( &tt1, NULL );
    JacobiGPU( a, n, m, .2, .1, .1, .1 );
    gettimeofday( &tt2, NULL );
    ms = (tt2.tv_sec - tt1.tv_sec);
    ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
    fms = (float)ms / 1000000.0f;
    printf( "time(gpu ) = %f seconds\n", fms );
}
