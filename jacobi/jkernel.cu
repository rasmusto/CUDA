
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

extern "C" __global__ void
jacobikernel( float* a, float* newa, float* lchange, int n, int m, float w0, float w1, float w2 )
{
    int ti = threadIdx.x;
    int tj = threadIdx.y;
    int i = blockIdx.x * blockDim.x + ti + 1;
    int j = blockIdx.y * blockDim.y + tj + 1;

    newa[j*m+i] = w0*a[j*m+i] +
	    w1 * (a[j*m+i-1] + a[(j-1)*m+i] +
		  a[j*m+i+1] + a[(j+1)*m+i]) +
	    w2 * (a[(j-1)*m+i-1] + a[(j+1)*m+i-1] +
		  a[(j-1)*m+i+1] + a[(j+1)*m+i+1]);

    __shared__ float mychange[256];
    int ii = ti+blockDim.x*tj;
    mychange[ii] = fabsf( newa[j*m+i] - a[j*m+i] );
    __syncthreads();
    int nn = blockDim.x * blockDim.y;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf( mychange[ii], mychange[ii+nn] );
	__syncthreads();
    }
    if( ii == 0 )
	lchange[blockIdx.x + gridDim.x*blockIdx.y] = mychange[0];
}

extern "C" __global__ void
reductionkernel( float* lchange, int n )
{
    __shared__ float mychange[256];
    float mych = 0.0f;
    int ii = threadIdx.x, m;
    if( ii < n ) mych = lchange[ii];
    m = blockDim.x;
    while( m <= n ){
	mych = fmaxf( mych, lchange[ii+m] );
	m += blockDim.x;
    }
    mychange[ii] = mych;
    __syncthreads();
    int nn = blockDim.x;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf(mychange[ii],mychange[ii+nn]);
	__syncthreads();
    }
    if( ii == 0 )
	lchange[0] = mychange[0];
}

static float JacobiIter( float* a, int n, int m, float w0, float w1, float w2 )
{
    int bx, by, gx, gy;
    size_t memsize;
    float change;
    bx = 16;
    by = 16;
    gx = (n-2)/bx;
    gy = (m-2)/by;
    float *da, *dnewa, *lchange;
    memsize = sizeof(float) * n * m;
    hipMalloc( &da, memsize );
    hipMalloc( &dnewa, memsize );
    hipMalloc( &lchange, gx * gy * sizeof(float) );

    dim3 block( bx, by );
    dim3 grid( gx, gy );
    hipMemcpy( da, a, memsize, hipMemcpyHostToDevice );
    hipMemcpy( dnewa, a, memsize, hipMemcpyHostToDevice );
    jacobikernel<<< grid, block >>>( da, dnewa, lchange, n, m, w0, w1, w2 );
    reductionkernel<<< 1, bx*by >>>( lchange, gx*gy );

    hipMemcpy( a, dnewa, memsize, hipMemcpyDeviceToHost );
    hipMemcpy( &change, lchange, sizeof(float), hipMemcpyDeviceToHost );
    hipFree( da );
    hipFree( dnewa );
    hipFree( lchange );
    return change;
}

extern "C"
void JacobiGPU( float* a, int n, int m, float w0, float w1, float w2, float tol )
{
    float change;
    int iters;


    iters = 0;
    do{
	++iters;
	change = JacobiIter( a, n, m, w0, w1, w2 );
    }while( change > tol );
    printf( "JacobiGPU  converged in %d iterations to residual %f\n", iters, change );
}
