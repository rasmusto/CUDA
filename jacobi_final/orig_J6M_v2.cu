
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

 __global__ void
jacobikernel( float* a, float* newa, float* lchange, int n, int m, int THR, float w0, float w1, float w2 )
{
    int ti = threadIdx.x;
    int tj = threadIdx.y;
    int i = blockIdx.x * blockDim.x + ti + 1;
    int j = blockIdx.y * blockDim.y + tj + 1;
    int TPT = THR + 2;
    int TMO = THR - 1;
    __shared__ float mychange[23*23];
    float mnewa, molda;


    mychange[tj*TPT+ti] = a[(j-1)*m+i-1];
    if( ti < 2 ) mychange[tj*TPT+ti+THR] = a[(j-1)*m+i+TMO];
    if( tj < 2 ) mychange[(tj+THR)*TPT+ti] = a[(j+TMO)*m+i-1];
    if( tj < 2 && ti < 2 ) mychange[(tj+THR)*TPT+ti+THR] = a[(j+TMO)*m+i+TMO];

    __syncthreads();

    molda = mychange[(tj+1)*TPT+(ti+1)];
    mnewa = w0*molda +
	    w1 * (mychange[(tj+1)*TPT+(ti  )] + mychange[(tj  )*TPT+(ti+1)] +
		  mychange[(tj+1)*TPT+(ti+2)] + mychange[(tj+2)*TPT+(ti+1)]) +
	    w2 * (mychange[(tj  )*TPT+(ti  )] + mychange[(tj+2)*TPT+(ti  )] +
		  mychange[(tj  )*TPT+(ti+2)] + mychange[(tj+2)*TPT+(ti+2)]);
    newa[j*m+i] = mnewa;
    __syncthreads();

    int ii = ti+blockDim.x*tj;
    mychange[ii] = fabsf( mnewa - molda );
    __syncthreads();
    int nn = blockDim.x * blockDim.y;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf( mychange[ii], mychange[ii+nn] );
	__syncthreads();
    }
    if( ii == 0 )
	lchange[blockIdx.x + gridDim.x*blockIdx.y] = mychange[0];
}

 __global__ void
reductionkernel( float* lchange, int n, int THR )
{
    __shared__ float mychange[23*23];
    float mych = 0.0f;
    int ii = threadIdx.x, m;
    if( ii < n ) mych = lchange[ii];
    m = blockDim.x;
    while( m <= n ){
	mych = fmaxf( mych, lchange[ii+m] );
	m += blockDim.x;
    }
    mychange[ii] = mych;
    __syncthreads();
    int nn = blockDim.x;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf(mychange[ii],mychange[ii+nn]);
	__syncthreads();
    }
    if( ii == 0 )
	lchange[0] = mychange[0];
}

static float sumtime;


void JacobiGPU( float* a, int n, int m, int numThr, float w0, float w1, float w2, float tol )
{
    float change;
    int iters;
    size_t memsize;
    int bx, by, gx, gy;
    float *da, *dnewa, *lchange;
    hipEvent_t e1, e2;

    bx = numThr;
    by = numThr;
    gx = (n-2)/bx + ((n-2)%bx == 0?0:1);
    gy = (m-2)/by + ((m-2)%by == 0?0:1);
    printf("Number of threads = %i and %i.\nNumber of Grids = %i and %i.\n", bx, by, gx, gy);

    sumtime = 0.0f;
    memsize = sizeof(float) * n * m;
    hipMalloc( &da, memsize );
    hipMalloc( &dnewa, memsize );
    hipMalloc( &lchange, gx * gy * sizeof(float) );
    hipEventCreate( &e1 );
    hipEventCreate( &e2 );

    dim3 block( bx, by );
    dim3 grid( gx, gy );

    iters = 0;
    hipMemcpy( da, a, memsize, hipMemcpyHostToDevice );
    hipMemcpy( dnewa, a, memsize, hipMemcpyHostToDevice );
    do{
	float msec;
	++iters;

	hipEventRecord( e1 );
	jacobikernel<<< grid, block >>>( da, dnewa, lchange, n, m, numThr, w0, w1, w2 );
	reductionkernel<<< 1, bx*by >>>( lchange, gx*gy, numThr );
	hipEventRecord( e2 );

	hipMemcpy( &change, lchange, sizeof(float), hipMemcpyDeviceToHost );
	hipEventElapsedTime( &msec, e1, e2 );
	sumtime += msec;
	float *ta;
	ta = da;
	da = dnewa;
	dnewa = ta; 
    }while( change > tol );
    printf( "JacobiGPU  converged in %d iterations to residual %f\n", iters, change );
    printf( "JacobiGPU  used %f seconds total\n", sumtime/1000.0f );
    hipMemcpy( a, dnewa, memsize, hipMemcpyDeviceToHost );
    hipFree( da );
    hipFree( dnewa );
    hipFree( lchange );
    hipEventDestroy( e1 );
    hipEventDestroy( e2 );
}

static void init( float* a, int n, int m )
{
    int i, j;
    memset( a, 0, sizeof(float) * n * m );
    /* boundary conditions */
    for( j = 0; j < n; ++j ){
	a[j*m+n-1] = j;
    }
    for( i = 0; i < m; ++i ){
	a[(n-1)*m+i] = i;
    }
    a[(n-1)*m+m-1] = m+n;
}

int main( int argc, char* argv[] )
{
    int n, m;
    float *a;
    struct timeval tt1, tt2;
    int ms;
    float fms;
    int numThr;

    if( argc <= 1 ){
	fprintf( stderr, "%s sizen [sizem]\n", argv[0] );
	return 1;
    }

    n = atoi( argv[1] );
    if( n <= 0 ) n = 100;
    m = n;
    if( argc > 3 ){
	m = atoi( argv[2] );
	numThr = atoi( argv[3] );
	if( m <= 0 ) m = 100;
    }

    printf( "Jacobi %d x %d\n", n, m );

    a = (float*)malloc( sizeof(float) * n * m );
    
    init( a, n, m );
 
    gettimeofday( &tt1, NULL );
    JacobiGPU( a, n, m, numThr, .2, .1, .1, .1 );
    gettimeofday( &tt2, NULL );
    ms = (tt2.tv_sec - tt1.tv_sec);
    ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
    fms = (float)ms / 1000000.0f;
    printf( "time(gpu ) = %f seconds\n", fms );
}
