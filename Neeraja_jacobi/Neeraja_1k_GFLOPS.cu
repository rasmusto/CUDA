
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

 __global__ void
jacobikernel( float* a, float* newa, float* lchange, int n, int m, float w0, float w1, float w2, int sz )
{
    int ti = threadIdx.x;
    int tj = threadIdx.y;
    int i = blockIdx.x * blockDim.x + ti + 1;
    int j = blockIdx.y * blockDim.y + tj + 1;
    __shared__ float mychange[18*18];
    float mnewa, molda;


    mychange[tj*18+ti] = a[(j-1)*m+(i-1)];
    if( ti < 2 ) mychange[tj*18+ti+16] = a[(j-1)*m+i+15];
    if( tj < 2 ) mychange[(tj+16)*18+ti] = a[(j+15)*m+i-1];
    if( tj < 2 && ti < 2 ) mychange[(tj+16)*18+ti+16] = a[(j+15)*m+i+15];

    __syncthreads();

    molda = mychange[(tj+1)*18+(ti+1)];
    mnewa = w0*molda +
	    w1 * (mychange[(tj+1)*18+(ti  )] + mychange[(tj  )*18+(ti+1)] +
		  mychange[(tj+1)*18+(ti+2)] + mychange[(tj+2)*18+(ti+1)]) +
	    w2 * (mychange[(tj  )*18+(ti  )] + mychange[(tj+2)*18+(ti  )] +
		  mychange[(tj  )*18+(ti+2)] + mychange[(tj+2)*18+(ti+2)]);
    newa[j*m+i] = mnewa;
    __syncthreads();

    int ii = ti+blockDim.x*tj;
    mychange[ii] = fabsf( mnewa - molda );
    __syncthreads();
    int nn = blockDim.x * blockDim.y;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf( mychange[ii], mychange[ii+nn] );
	__syncthreads();
    }
    
    if( ii == 0 )
	lchange[blockIdx.x + gridDim.x*blockIdx.y] = mychange[0];
	__syncthreads();
	
	float mych = 0.0f;
	
    int ni = ti+blockDim.x*tj;
 
    if( ni < sz ) mych = lchange[ni];
    
    int mm = 256;
    while( mm <= sz ){
	mych = fmaxf( mych, lchange[ni+mm] );
	mm += 256;
    }
    mychange[ni] = mych;
    __syncthreads();
    
    nn = blockDim.x*blockDim.x;
    while( (nn>>=1) > 0 ){
	if( ni < nn )
	    mychange[ni] = fmaxf(mychange[ni], mychange[ni+nn]);
	__syncthreads();
    }
    if( ni == 0 )
	lchange[0] = mychange[0];

}

static float sumtime;


void JacobiGPU( float* a, int n, int m, float w0, float w1, float w2, float tol )
{
    float change=0.0;
    int iters;
    size_t memsize;
    int bx, by, gx, gy;
    float *da, *dnewa, *lchange;
    hipEvent_t e1, e2;

    bx = 16;
    by = 16;
    gx = (n-2)/bx + ((n-2)%bx == 0?0:1);
    gy = (m-2)/by + ((m-2)%by == 0?0:1);

    sumtime = 0.0f;
    memsize = sizeof(float) * n * m;
    hipMalloc( &da, memsize );
    hipMalloc( &dnewa, memsize );
    hipMalloc( &lchange, gx * gy * sizeof(float) );
    hipEventCreate( &e1 );
    hipEventCreate( &e2 );

    dim3 block( bx, by );
    dim3 grid( gx, gy );

    iters = 0;
    hipMemcpy( da, a, memsize, hipMemcpyHostToDevice );
    hipMemcpy( dnewa, a, memsize, hipMemcpyHostToDevice );
    do{
	float msec;
	++iters;

	hipEventRecord( e1 );
	jacobikernel<<< grid, block >>>( da, dnewa, lchange, n, m, w0, w1, w2, gx*gy );
	hipEventRecord( e2 );

	hipMemcpy( &change, lchange, sizeof(float), hipMemcpyDeviceToHost );
	hipEventElapsedTime( &msec, e1, e2 );
	sumtime += msec;
	float *ta;
	ta = da;
	da = dnewa;
	dnewa = ta; 
    }while( change > tol );

    double time = sumtime/1000.0f;
    double dNumOps = 15 * iters * n * m;
    double gflops = dNumOps/time/1e9;

    printf( "JacobiGPU  converged in %d iterations to residual %f\n", iters, change );
    printf( "JacobiGPU  used %.5f seconds total\n", sumtime/1000.0f );
    printf( "Size(Number of Operations) = %.0f Ops/sec \n", dNumOps );
    printf( "Throughtput = %.4f GFlops/sec \n",gflops );

    
    hipMemcpy( a, dnewa, memsize, hipMemcpyDeviceToHost );
    hipFree( da );
    hipFree( dnewa );
    hipFree( lchange );
    hipEventDestroy( e1 );
    hipEventDestroy( e2 );
}

static void init( float* a, int n, int m )
{
    int i, j;
    memset( a, 0, sizeof(float) * n * m );
    /* boundary conditions */
    for( j = 0; j < n; ++j ){
	a[j*m+n-1] = j;
    }
    for( i = 0; i < m; ++i ){
	a[(n-1)*m+i] = i;
    }
    a[(n-1)*m+m-1] = m+n;
}

static void init1( float* a, int n, int m )
{
    int i, j;
    memset( a, 0, sizeof(float) * n * m );
    /* boundary conditions */
    for (i=0; i<m; i++)
     for (j=0; j<n; j++)
         a[i*n+j] = i;
}

int main( int argc, char* argv[] )
{
    int n, m;
    float *a;
    struct timeval tt1, tt2;
    int ms;
    float fms;


    if( argc <= 1 ){
	fprintf( stderr, "%s sizen [sizem]\n", argv[0] );
	return 1;
    }

    n = atoi( argv[1] );
    if( n <= 0 ) n = 100;
    m = n;
    if( argc > 2 ){
	m = atoi( argv[2] );
	if( m <= 0 ) m = 100;
    }

    printf( "Jacobi %d x %d\n", n, m );

    a = (float*)malloc( sizeof(float) * n * m );
    
    init( a, n, m );
 
    gettimeofday( &tt1, NULL );
    JacobiGPU( a, n, m, .2, .1, .1, .1 );
    gettimeofday( &tt2, NULL );
    ms = (tt2.tv_sec - tt1.tv_sec);
    ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
    fms = (float)ms / 1000000.0f;
    printf( "time(gpu ) = %f seconds\n", fms );
}
