
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

__global__ void
jacobikernel( float* a, float* newa, float* lchange, int n, int m, float w0, float w1, float w2 )
{
    int ti = threadIdx.x;
    int tj = threadIdx.y;
    int i = blockIdx.x * blockDim.x + ti + 1;
    int j = blockIdx.y * blockDim.y + tj + 1;

    newa[j*m+i] = w0*a[j*m+i] +
	    w1 * (a[j*m+i-1] + a[(j-1)*m+i] +
		  a[j*m+i+1] + a[(j+1)*m+i]) +
	    w2 * (a[(j-1)*m+i-1] + a[(j+1)*m+i-1] +
		  a[(j-1)*m+i+1] + a[(j+1)*m+i+1]);

    __shared__ float mychange[256];
    int ii = ti+blockDim.x*tj;
    mychange[ii] = fabsf( newa[j*m+i] - a[j*m+i] );
    __syncthreads();
    int nn = blockDim.x * blockDim.y;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf( mychange[ii], mychange[ii+nn] );
	__syncthreads();
    }
    if( ii == 0 )
	lchange[blockIdx.x + gridDim.x*blockIdx.y] = mychange[0];
}

 __global__ void
reductionkernel( float* lchange, int n )
{
    __shared__ float mychange[256];
    float mych = 0.0f;
    int ii = threadIdx.x, m;
    if( ii < n ) mych = lchange[ii];
    m = blockDim.x;
    while( m <= n ){
	mych = fmaxf( mych, lchange[ii+m] );
	m += blockDim.x;
    }
    mychange[ii] = mych;
    __syncthreads();
    int nn = blockDim.x;
    while( (nn>>=1) > 0 ){
	if( ii < nn )
	    mychange[ii] = fmaxf(mychange[ii],mychange[ii+nn]);
	__syncthreads();
    }
    if( ii == 0 )
	lchange[0] = mychange[0];
}

static float sumtime;


void JacobiGPU( float* a, int n, int m, float w0, float w1, float w2, float tol )
{
    float change;
    int iters;
    size_t memsize;
    int bx, by, gx, gy;
    float *da, *dnewa, *lchange;
    hipEvent_t e1, e2;

    bx = 16;
    by = 16;
    gx = (n-2)/bx;
    gy = (m-2)/by;

    sumtime = 0.0f;
    memsize = sizeof(float) * n * m;
    hipMalloc( &da, memsize );
    hipMalloc( &dnewa, memsize );
    hipMalloc( &lchange, gx * gy * sizeof(float) );
    hipEventCreate( &e1 );
    hipEventCreate( &e2 );

    dim3 block( bx, by );
    dim3 grid( gx, gy );

    iters = 0;
    hipMemcpy( da, a, memsize, hipMemcpyHostToDevice );
    hipMemcpy( dnewa, a, memsize, hipMemcpyHostToDevice );
    do{
	float msec;
	++iters;

	hipEventRecord( e1 );
	jacobikernel<<< grid, block >>>( da, dnewa, lchange, n, m, w0, w1, w2 );
	reductionkernel<<< 1, bx*by >>>( lchange, gx*gy );
	hipEventRecord( e2 );

	hipMemcpy( &change, lchange, sizeof(float), hipMemcpyDeviceToHost );
	hipEventElapsedTime( &msec, e1, e2 );
	sumtime += msec;
	float *ta;
	ta = da;
	da = dnewa;
	dnewa = ta; 
    }while( change > tol );
    printf( "JacobiGPU  converged in %d iterations to residual %f\n", iters, change );
    printf( "JacobiGPU  used %f seconds total\n", sumtime/1000.0f );
    hipMemcpy( a, dnewa, memsize, hipMemcpyDeviceToHost );
    hipFree( da );
    hipFree( dnewa );
    hipFree( lchange );
    hipEventDestroy( e1 );
    hipEventDestroy( e2 );
}

static void init( float* a, int n, int m )
{
    int i, j;
    memset( a, 0, sizeof(float) * n * m );
    /* boundary conditions */
    for( j = 0; j < n; ++j ){
	a[j*m+n-1] = j;
    }
    for( i = 0; i < m; ++i ){
	a[(n-1)*m+i] = i;
    }
    a[(n-1)*m+m-1] = m+n;
}

int
main( int argc, char* argv[] )
{
    int n, m;
    float *a;
    struct timeval tt1, tt2;
    int ms;
    float fms;

    if( argc <= 1 ){
	fprintf( stderr, "%s sizen [sizem]\n", argv[0] );
	return 1;
    }

    n = atoi( argv[1] );
    if( n <= 0 ) n = 100;
    m = n;
    if( argc > 2 ){
	m = atoi( argv[2] );
	if( m <= 0 ) m = 100;
    }

    printf( "Jacobi %d x %d\n", n, m );

    a = (float*)malloc( sizeof(float) * n * m );

    //init( a, n, m );

    gettimeofday( &tt1, NULL );
    //JacobiHost( a, n, m, .2, .1, .1, .1 );
    gettimeofday( &tt2, NULL );
    ms = (tt2.tv_sec - tt1.tv_sec);
    ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
    fms = (float)ms / 1000000.0f;
    //printf( "time(host) = %f seconds\n", fms );

    init( a, n, m );

    gettimeofday( &tt1, NULL );
    JacobiGPU( a, n, m, .2, .1, .1, .1 );
    gettimeofday( &tt2, NULL );
    ms = (tt2.tv_sec - tt1.tv_sec);
    ms = ms * 1000000 + (tt2.tv_usec - tt1.tv_usec);
    fms = (float)ms / 1000000.0f;
    printf( "time(gpu ) = %f seconds\n", fms );
}
